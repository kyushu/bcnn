
#include <hip/hip_runtime.h>
/*
* Copyright (c) 2016 Jean-Noel Braun.
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/


#ifdef BCNN_USE_CUDA

#include <bh/bh.h>

#include "bcnn/bcnn.h"
#include "bcnn_utils.h"

__global__ void _bcnn_forward_activation_layer_kernel(float *x, int sz, bcnn_activation a)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < sz) {
        switch (a) {
        case TANH:
            x[i] = (exp(2 * x[i]) - 1) / (exp(2 * x[i]) + 1);
            break;
        case RELU:
            x[i] = x[i] * (x[i] > 0);
            break;
        case RAMP:
            x[i] = x[i] * (x[i] > 0) + 0.1 * x[i];
            break;
        case CLAMP:
            x[i] = bh_clamp(x[i], 0, 1);
            break;
        case NONE:
            break;
        default:
            break;
        }
    }
    return;
}

int bcnn_forward_activation_gpu(float *x, int sz, bcnn_activation a)
{
    _bcnn_forward_activation_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(x,
        sz, a);
    return BCNN_SUCCESS;
}

int bcnn_forward_activation_layer_gpu(bcnn_layer *layer, bcnn_node *src_node, bcnn_node *dst_node)
{
    bcnn_tensor src = src_node->tensor;
    bcnn_tensor dst = dst_node->tensor;
    int sz = bcnn_tensor_get_size(&dst);

    dst.data_gpu = src.data_gpu;
    bcnn_forward_activation_gpu(dst.data_gpu, sz, layer->activation);
    bcnn_cuda_check(cudaPeekAtLastError());

    return BCNN_SUCCESS;
}


__global__ void _bcnn_backward_activation_layer_kernel(float *x, float *diff, int sz, bcnn_activation a)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < sz) {
        switch (a) {
        case TANH:
            diff[i] *= (1 - x[i] * x[i]);
            break;
        case RELU:
            diff[i] *= ((float)(x[i] > 0));
            break;
        case RAMP:
            diff[i] *= ((float)(x[i] > 0) + 0.1f);
            break;
        case CLAMP:
            diff[i] *= (float)(x[i] > 0.0f && (x[i] < 1.0f));
            break;
        case NONE:
            break;
        default:
            break;
        }
    }
}

int bcnn_backward_activation_gpu(float *x, float *dx, int sz, bcnn_activation a)
{
    _bcnn_backward_activation_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(x, dx, 
        sz, a);
    return BCNN_SUCCESS;
}

int bcnn_backward_activation_layer_gpu(bcnn_layer *layer, bcnn_node *src_node, bcnn_node *dst_node)
{
    bcnn_tensor src = src_node->tensor;
    bcnn_tensor dst = dst_node->tensor;
    int sz = bcnn_tensor_get_size(&dst);
    
    bcnn_backward_activation_gpu(dst.data_gpu, dst.grad_data_gpu, sz, layer->activation);
    bcnn_cuda_check(cudaPeekAtLastError());
    src.grad_data_gpu = dst.grad_data_gpu;

    return BCNN_SUCCESS;
}


#endif